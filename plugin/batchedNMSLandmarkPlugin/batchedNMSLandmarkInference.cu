/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "common/bboxUtils.h"
#include <hip/hip_runtime_api.h>
#include "common/kernel.h"
#include "common/nmsUtils.h"
#include "gatherNMSLandmarkOutputs.h"


pluginStatus_t nmsInferenceLandmark(hipStream_t stream,
                            const int N,
                            const int perBatchBoxesSize,
                            const int perBatchScoresSize,
                            const int perBatchLandmarksSize,
                            const bool shareLocation,
                            const int backgroundLabelId,
                            const int numPredsPerClass,
                            const int numClasses,
                            const int topK,
                            const int keepTopK,
                            const float scoreThreshold,
                            const float iouThreshold,
                            const nvinfer1::DataType DT_BBOX,
                            const void *locData,
                            const nvinfer1::DataType DT_SCORE,
                            const void *confData,
                            const void *landData,
                            void *keepCount,
                            void *nmsedBoxes,
                            void *nmsedScores,
                            void *nmsedClasses,
                            void *nmsedLandmarks,
                            void *workspace,
                            bool isNormalized,
                            bool confSigmoid,
                            bool clipBoxes,
                            int scoreBits,
                            bool caffeSemantics)
{
    // Calculate the total number of locations (batch size * number of boxes per sample * 4 coordinates per box)
    const int locCount = N * perBatchBoxesSize;

    /*
    * Determine the number of location classes:
    * If shareLocation is true, bounding boxes are shared among all classes (e.g., multi-class classification).
    * Otherwise, bounding boxes are specific to individual classes (binary classification).
    */
    const int numLocClasses = shareLocation ? 1 : numClasses;

    // Calculate the size of the bounding box data
    size_t bboxDataSize = detectionForwardBBoxDataSize(N, perBatchBoxesSize, DT_BBOX);

    // Allocate workspace for raw bounding box data
    void* bboxDataRaw = workspace;

    // Copy bounding box data from `locData` to `bboxDataRaw` on the GPU
    hipMemcpyAsync(bboxDataRaw, locData, bboxDataSize, hipMemcpyDeviceToDevice, stream);

    // Initialize plugin status variable
    pluginStatus_t status;

    /*
    * bboxDataRaw format:
    * [batch_size, numPriors (per sample), numLocClasses, 4 coordinates per box]
    */

    // Initialize bounding box data pointer
    void* bboxData;

    // Calculate the size required for permuted bounding box data
    size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, perBatchBoxesSize, DT_BBOX);

    // Allocate workspace for permuted bounding box data
    void* bboxPermute = nextWorkspacePtr(reinterpret_cast<int8_t*>(bboxDataRaw), bboxDataSize);


    /*
    * After permutation, bboxData format:
    * [batch_size, numLocClasses, numPriors (per sample) (numPredsPerClass), 4]
    * This is equivalent to swapping axis.
    */
    if (!shareLocation)
    {
        // Permute the bounding box data for non-shared locations
        status = permuteData(
            stream, locCount, numLocClasses, numPredsPerClass, 4, 
            DT_BBOX, /*applySigmoid=*/false, bboxDataRaw, bboxPermute);

        // Check the status and handle errors appropriately
        if (status != STATUS_SUCCESS)
        {
            return status; // Propagate the error for further handling
        }

        // Assign the permuted data to bboxData
        bboxData = bboxPermute;
    }
    else
    {
        /*
        * If shareLocation is true, numLocClasses = 1.
        * No need to permute data since it's already in linear memory format.
        */
        bboxData = bboxDataRaw;
    }

    /*
    * Conf data format:
    * [batch size, numPriors * param.numClasses, 1, 1]
    */
    const int numScores = N * perBatchScoresSize;

    // Calculate the total size of scores before NMS
    size_t totalScoresSize = detectionForwardPreNMSSize(N, perBatchScoresSize);

    // If the data type is half-precision, adjust the size accordingly
    if (DT_SCORE == nvinfer1::DataType::kHALF)
    {
        totalScoresSize /= 2; // detectionForwardPreNMSSize assumes kFLOAT
    }

    // Allocate workspace memory for scores
    void* scores = nextWorkspacePtr(reinterpret_cast<int8_t*>(bboxPermute), bboxPermuteSize);

    // Perform permutation on confidence scores
    /*
    * After permutation, scores format:
    * [batch_size, numClasses, numPredsPerClass, 1]
    */
    status = permuteData(
        stream, numScores, numClasses, numPredsPerClass, 1, 
        DT_SCORE, confSigmoid, confData, scores);

    // Check the status of permutation and handle errors
    if (status != STATUS_SUCCESS)
    {
        return status; // Propagate the error
    }

    // Calculate the size of indices needed for NMS
    size_t indicesSize = detectionForwardPreNMSSize(N, perBatchScoresSize);

    // Allocate workspace memory for indices
    void* indices = nextWorkspacePtr(reinterpret_cast<int8_t*>(scores), totalScoresSize);

    // Calculate the size of post-NMS scores
    size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK);

    // Adjust size if using half-precision data type
    if (DT_SCORE == nvinfer1::DataType::kHALF)
    {
        postNMSScoresSize /= 2; // detectionForwardPostNMSSize assumes kFLOAT
    }

    // Calculate the size of post-NMS indices
    size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK); // indices are int32

    // Allocate workspace for post-NMS scores
    void* postNMSScores = nextWorkspacePtr(reinterpret_cast<int8_t*>(indices), indicesSize);

    // Allocate workspace for post-NMS indices
    void* postNMSIndices = nextWorkspacePtr(reinterpret_cast<int8_t*>(postNMSScores), postNMSScoresSize);

    // Allocate workspace for sorting
    void* sortingWorkspace = nextWorkspacePtr(reinterpret_cast<int8_t*>(postNMSIndices), postNMSIndicesSize);

    // Handle score shift if using half-precision and scoreBits are within a specific range
    float scoreShift = 0.f;
    if (DT_SCORE == nvinfer1::DataType::kHALF && scoreBits > 0 && scoreBits <= 10)
    {
        scoreShift = 1.f;
    }

    // Sort scores per class so NMS can be applied
    status = sortScoresPerClass(
        stream, N, numClasses, numPredsPerClass, backgroundLabelId, 
        scoreThreshold, DT_SCORE, scores, indices, sortingWorkspace, 
        scoreBits, scoreShift);

    // Check for errors in sorting
    if (status != STATUS_SUCCESS)
    {
        return status; // Propagate error
    }


    // The bounding boxes are in the format [ymin, xmin, ymax, xmax].
    // FlipXY is set to true as the default implementation assumes [xmin, ymin, xmax, ymax].
    bool flipXY = true;

    // Perform Non-Maximum Suppression (NMS)
    status = allClassNMS(
        stream, N, numClasses, numPredsPerClass, topK, iouThreshold, 
        shareLocation, isNormalized, DT_SCORE, DT_BBOX, bboxData, 
        scores, indices, postNMSScores, postNMSIndices, flipXY, 
        scoreShift, caffeSemantics);

    // Check the status of NMS and handle errors
    if (status != STATUS_SUCCESS)
    {
        return status; // Propagate the error
    }

    // Sort the bounding boxes after NMS using scores
    status = sortScoresPerImage(
        stream, N, numClasses * topK, DT_SCORE, postNMSScores, 
        postNMSIndices, scores, indices, sortingWorkspace, scoreBits);

    // Check the status of sorting and handle errors
    if (status != STATUS_SUCCESS)
    {
        return status; // Propagate the error
    }

    // Gather data from the sorted bounding boxes after NMS
    status = gatherNMSLandmarkOutputs(
        stream, shareLocation, N, numPredsPerClass, numClasses, topK, 
        keepTopK, DT_BBOX, DT_SCORE, indices, scores, bboxData, 
        landData, keepCount, nmsedBoxes, nmsedScores, nmsedClasses, 
        nmsedLandmarks, clipBoxes, scoreShift);

    // Check the status of data gathering and handle errors
    if (status != STATUS_SUCCESS)
    {
        return status; // Propagate the error
    }

    // Return success status
    return STATUS_SUCCESS;

}

